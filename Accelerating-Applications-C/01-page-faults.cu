
#include <hip/hip_runtime.h>

__global__ void deviceKernel(int *a, int N)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = idx; i < N; i += stride)
  {
    a[i] = 1;
  }
}

void hostFunction(int *a, int N)
{
  for (int i = 0; i < N; ++i)
  {
    a[i] = 1;
  }
}

int main()
{

  int N = 2<<24;
  size_t size = N * sizeof(int);
  int *a;
  hipMallocManaged(&a, size);

  /*
   * Conduct experiments to learn more about the behavior of
   * `cudaMallocManaged`.
   *
   * What happens when unified memory is accessed only by the GPU? 241 page faults

    deviceKernel<<<256, 256>>>(a, N);
    cudaDeviceSynchronize();
    cudaFree(a);

   * What happens when unified memory is accessed only by the CPU? 384 page faults

      hostFunction(a, N);
      cudaFree(a);

   * What happens when unified memory is accessed first by the GPU then the CPU? 234 page faults

    deviceKernel<<<256, 256>>>(a, N);
    cudaDeviceSynchronize();
    hostFunction(a, N);
    cudaFree(a);

   * What happens when unified memory is accessed first by the CPU then the GPU? 384 page faults

    hostFunction(a, N);
    deviceKernel<<<256, 256>>>(a, N);
    cudaDeviceSynchronize();
    cudaFree(a);
   * Hypothesize about UM behavior, page faulting specificially, before each
   * experiement, and then verify by running `nvprof`.
   */
  deviceKernel<<<256, 256>>>(a, N);
  hipDeviceSynchronize();
  hostFunction(a,N);
  hipFree(a);
}