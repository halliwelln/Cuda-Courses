
#include <hip/hip_runtime.h>
#include <stdio.h>


/*
Print:
'Hello from GPU'
'Hello from CPU'
'Hello from GPU'
*/
void helloCPU()
{
  printf("Hello from the CPU.\n");
}


__global__ void helloGPU()
{
  printf("Hello also from the GPU.\n");
}

int main()
{

  helloGPU<<<1,1>>>();

  hipDeviceSynchronize();

  helloCPU();

  helloGPU<<<1,1>>>();

  hipDeviceSynchronize();


}
