
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
  'Loop from 0 to 9'
 */

__global__ void loop()
{
    printf("This is iteration number %d\n", threadIdx.x);
}

int main()
{
  /*
   * When refactoring `loop` to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * For this exercise, only use 1 block of threads.
   */
  loop<<<1,10>>>();
  
  hipDeviceSynchronize();
}
