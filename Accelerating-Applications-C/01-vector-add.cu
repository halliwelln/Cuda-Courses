
#include <hip/hip_runtime.h>
#include <stdio.h>

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = gridDim.x * blockDim.x;
  
  for(int i = idx; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipError_t sync_err;
  hipError_t async_err;
  
  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);
  
  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);
  
  int threadsPerBlock = 256;
  int numberOfBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;
  
  addVectorsInto<<<numberOfBlocks,threadsPerBlock>>>(c, a, b, N);
  
  sync_err = hipGetLastError();
  
  async_err = hipDeviceSynchronize();
  
  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}