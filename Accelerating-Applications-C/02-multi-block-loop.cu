
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */

__global__ void loop()
{
printf("This is iteration number %d\n", threadIdx.x + blockIdx.x * blockDim.x);
}

int main()
{
  /*
   * When refactoring `loop` to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * For this exercise, be sure to use at least 2 blocks in
   * the execution configuration.
   */

  loop<<<2,5>>>();
  
  hipDeviceSynchronize();
}
