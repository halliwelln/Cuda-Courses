
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__
void doubleElements(int *a, int N)
{

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  for (int i = idx; i < N + stride; i += stride)
  {
    a[i] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  /*
   * Add error handling to this source code to learn what errors
   * exist, and then correct them. Googling error messages may be
   * of service if actions for resolving them are not clear to you.
   */

  int N = 10000;
  int *a;
  hipError_t sync_err;
  hipError_t async_err;
  
  size_t size = N * sizeof(int);
  hipMallocManaged(&a, size);

  init(a, N);

  //size_t threads_per_block = 2048;
  size_t threads_per_block = 1024;
  size_t number_of_blocks = 32;

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  
  sync_err = hipGetLastError();
  async_err = hipDeviceSynchronize();
  
  if (async_err != hipSuccess)
  {
  printf("Sync error: %s\n", hipGetErrorString(async_err));
  }

  if (sync_err != hipSuccess)
  {
  printf("Device error: %s\n", hipGetErrorString(sync_err));
  }


  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}