
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
Print:
'Hello from the CPU.'
'Hello from the GPU.'
*/

void helloCPU()
{
  printf("Hello from the CPU.\n");
}


__global__ void helloGPU()
{
  printf("Hello also from the GPU.\n");
}

int main()
{

  helloCPU();

  helloGPU<<<1,1>>>();

  hipDeviceSynchronize();
}
