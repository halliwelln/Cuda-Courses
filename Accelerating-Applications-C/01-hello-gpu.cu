
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
Print:
'Hello from the GPU.'
'Hello from the CPU.'
*/

void helloCPU()
{
  printf("Hello from the CPU.\n");
}


__global__ void helloGPU()
{
  printf("Hello from the GPU.\n");
}

int main()
{

  helloGPU<<<1,1>>>();

  hipDeviceSynchronize();

  helloCPU();
}
